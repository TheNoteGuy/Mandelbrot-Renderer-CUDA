#include "hip/hip_runtime.h"
extern "C" {
    __global__ void mandelbrotKernel(int* result, int width, int height, decimal XMin, decimal YMin, decimal xScale, decimal yScale, int MaxIterations, decimal brightness, int redPart, int greenPart, int bluePart) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
            decimal zx = (decimal)0.0, zy = (decimal)0.0;
            decimal cx = XMin + x * xScale;
            decimal cy = YMin + y * yScale;
            int iterations = 0;

            while (iterations < MaxIterations && zx * zx + zy * zy < (decimal)4.0) {
                decimal temp = zx * zx - zy * zy + cx;
                zy = (decimal)2.0 * zx * zy + cy;
                zx = temp;
                iterations++;
            }

            if (iterations == MaxIterations)
                result[y * width + x] = MaxIterations;
            else
                result[y * width + x] = iterations;
        }
    }
}