#include "hip/hip_runtime.h"
#include "cgbn.h"
#include "gpu_support.h"

extern "C" {
    __global__ void mandelbrotKernel(int* result, int width, int height, double XMin, double YMin, double xScale, double yScale, int MaxIterations, double brightness, int redPart, int greenPart, int bluePart) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
            double zx = 0.0, zy = 0.0;
            double cx = XMin + x * xScale;
            double cy = YMin + y * yScale;
            int iterations = 0;

            while (iterations < MaxIterations && zx * zx + zy * zy < 4.0) {
                double temp = zx * zx - zy * zy + cx;
                zy = 2.0 * zx * zy + cy;
                zx = temp;
                iterations++;
            }

            if (iterations == MaxIterations)
                result[y * width + x] = 0; // Change this value based on your color scheme
            else
                result[y * width + x] = iterations;
        }
    }
}